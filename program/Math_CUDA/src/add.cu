#include "hip/hip_runtime.h"
#include "add.cuh"

__global__ void add(int a, int b, int *c) //kernel function，running on gpu  
{  
*c = a + b;  
}  


int add(int a,int b)
{  
    int c;  
    int *dev_c;  
    hipMalloc((void**)&dev_c, sizeof(int)); 
    add<<<1,1>>>(a, b, dev_c);  
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);                                                               
    hipFree(dev_c); 
    return c;  
}  

int TEST::ADD()
{  
    int c;  
    int *dev_c;  
    hipMalloc((void**)&dev_c, sizeof(int));  
    add<<<1,1>>>(a, b, dev_c);  
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);                                                             
    hipFree(dev_c);  
    return c;  
}  
